#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#define GROUPSIZE 2

using namespace std;
__device__ int mandel(float c_re, float c_im, int maxIteration)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int* device_memo, int resX, int resY, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    for(int i=0; i<GROUPSIZE; i++)
		for(int j=0; j<GROUPSIZE; j++){
			int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUPSIZE + i;
			int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * GROUPSIZE + j;
			if(thisX >= resX || thisY >= resY) return ;
			float x = lowerX + thisX * stepX;
			float y = lowerY + thisY * stepY;
			device_memo[thisY * resX + thisX] = mandel(x, y, maxIterations);
		}
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
 
	//20 / 1024
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, mandelKernel, 0, resX * resY); 	
	//cout << width << minGridSize << " " << blockSize << endl;
	//
	int blockSizeSqrt = (int) sqrt(blockSize);
	int width = -blockSizeSqrt & blockSizeSqrt;

	int gridX = (resX + (width * GROUPSIZE) - 1) / (width * GROUPSIZE);
	int gridY = (resY + (width * GROUPSIZE) - 1) / (width * GROUPSIZE);

	dim3 block(width, width);
	dim3 grid(gridX, gridY);

	int* host_memo;
	int* device_memo;
	size_t pitch;
	hipMallocPitch((void **)&device_memo, &pitch, resX * sizeof(int), resY);
	hipHostAlloc((void **)&host_memo, resX * resY * sizeof(int), hipHostMallocDefault);

	mandelKernel <<< grid, block >>> (lowerX, lowerY, stepX, stepY, device_memo, resX, resY, maxIterations);
	hipDeviceSynchronize();
	hipMemcpy(host_memo, device_memo, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(device_memo);

	hipMemcpy(img, host_memo, resX * resY * sizeof(int), hipMemcpyHostToHost);
	hipFree(host_memo);
}
